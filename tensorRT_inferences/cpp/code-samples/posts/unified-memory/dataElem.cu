/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime.h>
#include <string.h>
#include <stdio.h>

struct DataElement
{
  char *name;
  int value;
};

__global__ 
void Kernel(DataElement *elem) {
  printf("On device: name=%s, value=%d\n", elem->name, elem->value);

  elem->name[0] = 'd';
  elem->value++;
}

void launch(DataElement *elem) {
  DataElement *d_elem;
  char *d_name;

  int namelen = strlen(elem->name) + 1;

  // Allocate storage for struct and text
  hipMalloc(&d_elem, sizeof(DataElement));
  hipMalloc(&d_name, namelen);

  // Copy up each piece separately, including new “text” pointer value
  hipMemcpy(d_elem, elem, sizeof(DataElement), hipMemcpyHostToDevice);
  hipMemcpy(d_name, elem->name, namelen, hipMemcpyHostToDevice);
  hipMemcpy(&(d_elem->name), &d_name, sizeof(char*), hipMemcpyHostToDevice);

  // Finally we can launch our kernel, but CPU & GPU use different copies of “elem”
  Kernel<<< 1, 1 >>>(d_elem);

  hipMemcpy(&(elem->value), &(d_elem->value), sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(elem->name, d_name, namelen, hipMemcpyDeviceToHost);

  hipFree(d_name);
  hipFree(d_elem);
}

int main(void)
{
  DataElement *e;
  e = (DataElement*)malloc(sizeof(DataElement));

  e->value = 10;
  e->name = (char*)malloc(sizeof(char) * (strlen("hello") + 1));
  strcpy(e->name, "hello");

  launch(e);

  printf("On host: name=%s, value=%d\n", e->name, e->value);

  free(e->name);
  free(e);

  hipDeviceReset();
}