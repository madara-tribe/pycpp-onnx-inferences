/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <hip/hip_runtime.h>
#include <string.h>
#include <stdio.h>

class Managed 
{
public:
  void *operator new(size_t len) {
    void *ptr;
    hipMallocManaged(&ptr, len);
    hipDeviceSynchronize();
    return ptr;
  }

  void operator delete(void *ptr) {
    hipDeviceSynchronize();
    hipFree(ptr);
  }
};

struct DataElement : public Managed
{
  char *name;
  int value;
};

__global__ 
void Kernel(DataElement *elem) {
  printf("On device: name=%s, value=%d\n", elem->name, elem->value);

  elem->name[0] = 'd';
  elem->value++;
}

void launch(DataElement *elem) {
  Kernel<<< 1, 1 >>>(elem);
  hipDeviceSynchronize();
}

int main(void)
{
  DataElement *e = new DataElement;
  
  e->value = 10;
  hipMallocManaged((void**)&(e->name), sizeof(char) * (strlen("hello") + 1) );
  strcpy(e->name, "hello");

  launch(e);

  printf("On host: name=%s, value=%d\n", e->name, e->value);

  hipFree(e->name);
  delete e;

  hipDeviceReset();
}

