/* Copyright (c) 2014, NVIDIA CORPORATION. All rights reserved.
 * 
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */


#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void init_data_kernel( int n, double* x)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if ( i < n )
	{
		x[i] = n - i;
	}
}


__global__ void daxpy_kernel(int n, double a, double * x, double * y)
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n)
	{
		y[i] = a*x[i] + y[i];
	}
}

__global__ void check_results_kernel( int n, double correctvalue, double * x )
{
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if (i < n)
	{
		if ( x[i] != correctvalue )
		{
			printf("ERROR at index = %d, expected = %f, actual: %f\n",i,correctvalue,x[i]);
		}
	}
}

void init_host_data( int n, double * x )
{
	for (int i=0; i<n; ++i)
	{
		x[i] = i;
	}
}

void init_data(int n, double* x, double* x_d, double* y_d)
{
	hipStream_t copy_stream;
	hipStream_t compute_stream;
	hipStreamCreate(&copy_stream);
	hipStreamCreate(&compute_stream);

	hipMemcpyAsync( x_d, x, n*sizeof(double), hipMemcpyDefault, copy_stream );
	init_data_kernel<<<ceil(n/256),256,0,compute_stream>>>(n, y_d);

	hipStreamSynchronize(copy_stream);
	hipStreamSynchronize(compute_stream);

	hipStreamDestroy(compute_stream);
	hipStreamDestroy(copy_stream);
}

void daxpy(int n, double a, double* x_d, double* y_d)
{
	daxpy_kernel<<<ceil(n/256),256>>>(n,a,x_d,y_d);
	hipDeviceSynchronize();
}

void check_results( int n, double correctvalue, double* x_d )
{
	check_results_kernel<<<ceil(n/256),256>>>(n,correctvalue,x_d);
}

void run_test(int n)
{
	double* x;
	double* x_d;
	double* y_d;
	hipSetDevice(0);
	hipHostMalloc((void**) &x, n*sizeof(double), hipHostMallocDefault);
	hipMalloc((void**)&x_d,n*sizeof(double));
	hipMalloc((void**)&y_d,n*sizeof(double));

	init_host_data(n, x);

	init_data(n,x,x_d,y_d);

	daxpy(n,1.0,x_d,y_d);

	check_results(n, n, y_d);

	hipFree(y_d);
	hipFree(x_d);
	hipHostFree(x);
	hipDeviceSynchronize();
}

int main()
{
	int n = 1<<22;
	run_test(n);
	return 0;
}
