#include "hip/hip_runtime.h"
/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
#include <cstdio>
#include "device_reduce_atomic.h"
#include "device_reduce_block_atomic.h"
#include "device_reduce_warp_atomic.h"
#include "device_reduce_stable.h"
#include "vector_functions.h"
#include "cub/hipcub/hipcub.hpp"

#define cudaCheckError() {                                          \
  hipError_t e=hipGetLastError();                                  \
  if(e!=hipSuccess) {                                               \
  printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));           \
  exit(0); \
  }                                                                  \
}

void RunTest(char* label, void (*fptr)(int* in, int* out, int N), int N, int REPEAT, int* src, int checksum) {
  int *in, *out;
  
  //allocate a buffer that is at least large enough that we can ensure it doesn't just sit in l2.
  int MIN_SIZE=4*1024*1024;
  int size=max(int(sizeof(int)*N),MIN_SIZE);
  
  //compute mod base for picking the correct buffer
  int mod=size/(N*sizeof(int));
  hipEvent_t start,stop;
  hipMalloc(&in,size);
  hipMalloc(&out,sizeof(int)*1024);  //only stable version needs multiple elements, all others only need 1
  hipEventCreate(&start);
  hipEventCreate(&stop);
  cudaCheckError();

  hipMemcpy(in,src,N*sizeof(int),hipMemcpyHostToDevice);
  
  //warm up
  fptr(in,out,N);

  hipDeviceSynchronize();
  cudaCheckError();
  hipEventRecord(start);

  for(int i=0;i<REPEAT;i++) {
    //iterate through different buffers
    int o=i%mod;
    fptr(in+o*N,out,N);
  }
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();

  float time_ms;
  hipEventElapsedTime(&time_ms,start,stop);
  float time_s=time_ms/(float)1e3;

  float GB=(float)N*sizeof(int)*REPEAT;
  float GBs=GB/time_s/(float)1e9;

  int sum;
  hipMemcpy(&sum,out,sizeof(int),hipMemcpyDeviceToHost);
  cudaCheckError();

  char *valid;
  if(sum==checksum) 
    valid="CORRECT";
  else
    valid="INCORRECT";

  printf("%s: %s, Time: %f s, GB/s: %f\n", label, valid, time_s, GBs); 
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(in);
  hipFree(out);
  cudaCheckError();
}

void RunTestCub(char* label, int N, int REPEAT, int* src, int checksum) {
  int *in, *out;
  hipEvent_t start,stop;
  
  hipMalloc(&in,sizeof(int)*N);
  hipMalloc(&out,sizeof(int)*1024);  //only stable version needs multiple elements, all others only need 1
  hipEventCreate(&start);
  hipEventCreate(&stop);
  cudaCheckError();

  hipMemcpy(in,src,N*sizeof(int),hipMemcpyHostToDevice);

  size_t temp_storage_bytes;
  int* temp_storage=NULL;
  hipcub::DeviceReduce::Reduce(temp_storage, temp_storage_bytes, in, out, N, hipcub::Sum());
  hipMalloc(&temp_storage,temp_storage_bytes);

  hipDeviceSynchronize();
  cudaCheckError();
  hipEventRecord(start);

  for(int i=0;i<REPEAT;i++) {
    hipcub::DeviceReduce::Reduce(temp_storage, temp_storage_bytes, in, out, N, hipcub::Sum());
  }
  hipEventRecord(stop);
  hipDeviceSynchronize();
  cudaCheckError();

  float time_ms;
  hipEventElapsedTime(&time_ms,start,stop);
  float time_s=time_ms/(float)1e3;

  float GB=(float)N*sizeof(int)*REPEAT;
  float GBs=GB/time_s/(float)1e9;

  int sum;
  hipMemcpy(&sum,out,sizeof(int),hipMemcpyDeviceToHost);
  cudaCheckError();

  char *valid;
  if(sum==checksum) 
    valid="CORRECT";
  else
    valid="INCORRECT";

  printf("%s: %s, Time: %f s, GB/s: %f\n", label, valid, time_s, GBs); 
  hipEventDestroy(start);
  hipEventDestroy(stop);
  hipFree(in);
  hipFree(out);
  hipFree(temp_storage);
  cudaCheckError();
}

int main(int argc, char** argv)
{
  if(argc!=3) {
    printf("Usage: ./reduce num_elems repeat\n");
    exit(0);
  }
  int NUM_ELEMS=atoi(argv[1]);
  int REPEAT=atoi(argv[2]);

  printf("NUM_ELEMS: %d, REPEAT: %d\n", NUM_ELEMS, REPEAT);

  int* vals=(int*)malloc(NUM_ELEMS*sizeof(int));
  int checksum =0;
  for(int i=0;i<NUM_ELEMS;i++) {
    vals[i]=rand()%4;
    checksum+=vals[i];
  }

  RunTest("device_reduce_atomic", device_reduce_atomic,NUM_ELEMS,REPEAT,vals,checksum);
  RunTest("device_reduce_atomic_vector2", device_reduce_atomic_vector2,NUM_ELEMS,REPEAT,vals,checksum);
  //RunTest("device_reduce_atomic_vector4", device_reduce_atomic_vector4,NUM_ELEMS,REPEAT,vals,checksum);
  
  RunTest("device_reduce_warp_atomic",device_reduce_warp_atomic,NUM_ELEMS,REPEAT,vals,checksum);
  RunTest("device_reduce_warp_atomic_vector2",device_reduce_warp_atomic_vector2,NUM_ELEMS,REPEAT,vals,checksum);
  //RunTest("device_reduce_warp_atomic_vector4",device_reduce_warp_atomic_vector4,NUM_ELEMS,REPEAT,vals,checksum);
  
  RunTest("device_reduce_block_atomic",device_reduce_block_atomic,NUM_ELEMS,REPEAT,vals,checksum);
  RunTest("device_reduce_block_atomic_vector2",device_reduce_block_atomic_vector2,NUM_ELEMS,REPEAT,vals,checksum);
  //RunTest("device_reduce_block_atomic_vector4",device_reduce_block_atomic_vector4,NUM_ELEMS,REPEAT,vals,checksum);
  
  RunTest("device_reduce_stable",device_reduce_stable,NUM_ELEMS,REPEAT,vals,checksum);
  RunTest("device_reduce_stable_vector2",device_reduce_stable_vector2,NUM_ELEMS,REPEAT,vals,checksum);
  //RunTest("device_reduce_stable_vector4",device_reduce_stable_vector4,NUM_ELEMS,REPEAT,vals,checksum);

  RunTestCub("device_reduce_cub",NUM_ELEMS,REPEAT,vals,checksum);
  
  free(vals);

}
