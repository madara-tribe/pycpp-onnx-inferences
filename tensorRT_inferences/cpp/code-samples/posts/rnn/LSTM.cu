#include "hip/hip_runtime.h"
/* Copyright (c) 1993-2016, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */
 
 
/*
 Compile:
  nvcc -arch=sm_52 -O3 -lcublas -lcurand -o LSTM LSTM.cu 
  
  To enable/disable different performance options add the flat -DPERFOPTSx
  Where x is a bitmask defining the options used (see below).
  
 Run:
  ./LSTM
  or
  ./LSTM <seqLength> <numLayers> <hiddenSize> <miniBatch>
  
 Example (run on an NVIDIA M40):
   > ./LSTM
   Running with default settings
   seqLength 100, numLayers 4, hiddenSize 512, miniBatch 64
   i checksum (example 0) 5.113463E+04
   h checksum (example 0) 2.048000E+03
   c checksum (example 0) 2.058137E+05
   i checksum 3.272639E+06     c checksum 1.317278E+07     h checksum 1.310720E+05
   Runtime 27.807743ms
*/

#include <stdio.h>
#include <hipblas.h>
#include <hiprand.h>

// Performance is not significantly different, but false saves memory. 
// False does not work with unfused pointwise ops.
#define TRAINING (false)

#ifndef PERFOPTS
   #define PERFOPTS (31)
#endif

#define GROUP_GEMM ((PERFOPTS & 1))
#define USE_STREAMS ((PERFOPTS & 2))
#define FUSE_PW ((PERFOPTS & 4))
#define PRE_TRANSPOSE ((PERFOPTS & 8))
#define RECUR_BATCH_SIZE (((PERFOPTS & 16) ? 2 : 1))

// Define some error checking macros.
#define cudaErrCheck(stat) { cudaErrCheck_((stat), __FILE__, __LINE__); }
void cudaErrCheck_(hipError_t stat, const char *file, int line) {
   if (stat != hipSuccess) {
      fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(stat), file, line);
   }
}

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
   if (stat != HIPBLAS_STATUS_SUCCESS) {
      fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
   }
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
   if (stat != HIPRAND_STATUS_SUCCESS) {
      fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
   }
}


// Device functions
__forceinline__ __device__ float sigmoidf(float in) {
   return 1.f / (1.f + expf(-in));  
}

// Pointwise functions
__global__ void pw_biasAdd(float *y, float *bias, int n, int nBias) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < n) y[i] += bias[i % nBias];
}

__global__ void pw_vecAdd(float *y, float *a,  float *b, int n) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < n) y[i] = a[i] + b[i];
}

__global__ void pw_vecMul(float *y, float *a,  float *b, int n) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < n) y[i] = a[i] * b[i];
}

__global__ void pw_tanh(float *y, float *a, int n) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < n) y[i] = tanh(a[i]);
}

__global__ void pw_sigmoid(float *y, float *a, int n) {
   int i = blockIdx.x * blockDim.x + threadIdx.x;
   if (i < n) y[i] = sigmoidf(a[i]);
}

// Unfused LSTM (calling many pointwise kernels).
int LSTM_elementwise_unfused( int hiddenSize, 
                               int miniBatch,
                               float * __restrict__ tmp_h, 
                               float * __restrict__ tmp_i, 
                               float * __restrict__ bias,
                               float * __restrict__ linearGates,
                               float * __restrict__ h_data,
                               float * __restrict__ i_data,
                               float * __restrict__ c_in,
                               float * __restrict__ c_out,
                               bool training,
                               hipStream_t stream) {
   dim3 blockDim;
   dim3 gridDim;
   
   int numElements = hiddenSize * miniBatch;
   
   blockDim.x = 128;
   gridDim.x = (numElements + blockDim.x - 1) / blockDim.x;

                 
   for (int i = 0; i < 4; i++) {
      if (tmp_h != NULL) {
         pw_vecAdd <<< gridDim, blockDim, 0, stream >>> (tmp_i + i * numElements, tmp_i  + i * numElements, tmp_h  + i * numElements, numElements);
         cudaErrCheck(hipGetLastError());
      }

      pw_biasAdd <<< gridDim, blockDim, 0, stream >>> (tmp_i + i * numElements, bias + i       * hiddenSize, numElements, hiddenSize);
      cudaErrCheck(hipGetLastError());
      
      pw_biasAdd <<< gridDim, blockDim, 0, stream >>> (tmp_i + i * numElements, bias + (i + 4) * hiddenSize, numElements, hiddenSize);
      cudaErrCheck(hipGetLastError());
      
      if (training) {
         printf("LSTM_elementWise_unfused does not support training\n"); 
         return 1;
      }
   }   
   
   pw_sigmoid <<< gridDim, blockDim, 0, stream >>> (tmp_i + 0 * numElements, tmp_i + 0 * numElements, numElements);
   cudaErrCheck(hipGetLastError());
   
   pw_sigmoid <<< gridDim, blockDim, 0, stream >>> (tmp_i + 1 * numElements, tmp_i + 1 * numElements, numElements);
   cudaErrCheck(hipGetLastError());
   
   pw_tanh    <<< gridDim, blockDim, 0, stream >>> (tmp_i + 2 * numElements, tmp_i + 2 * numElements, numElements);
   cudaErrCheck(hipGetLastError());
   
   pw_sigmoid <<< gridDim, blockDim, 0, stream >>> (tmp_i + 3 * numElements, tmp_i + 3 * numElements, numElements);
   cudaErrCheck(hipGetLastError());
   
   float *in_gate     = tmp_i + 0 * numElements;
   float *forget_gate = tmp_i + 1 * numElements;
   float *in_gate2    = tmp_i + 2 * numElements;
   float *out_gate    = tmp_i + 3 * numElements;
   
   if (c_in == NULL) {
      pw_vecMul <<< gridDim, blockDim, 0, stream >>> (in_gate, in_gate, in_gate2, numElements);
      cudaErrCheck(hipGetLastError());
   }
   else {              
      pw_vecMul <<< gridDim, blockDim, 0, stream >>> (forget_gate, forget_gate, c_in, numElements);
      cudaErrCheck(hipGetLastError());
      
      pw_vecMul <<< gridDim, blockDim, 0, stream >>> (in_gate, in_gate, in_gate2, numElements);
      cudaErrCheck(hipGetLastError());
      
      pw_vecAdd <<< gridDim, blockDim, 0, stream >>> (in_gate, in_gate, forget_gate, numElements);
      cudaErrCheck(hipGetLastError());
   }
   
   if (c_out != NULL) {
      cudaErrCheck(hipMemcpyAsync(c_out, in_gate, numElements * sizeof(float), hipMemcpyDeviceToDevice, stream));
   }
   
   pw_tanh <<< gridDim, blockDim, 0, stream >>> (in_gate, in_gate, numElements);
   cudaErrCheck(hipGetLastError());
   
   pw_vecMul <<< gridDim, blockDim, 0, stream >>> (h_data, out_gate, in_gate, numElements);
   cudaErrCheck(hipGetLastError());
   
   pw_vecMul <<< gridDim, blockDim, 0, stream >>> (i_data, out_gate, in_gate, numElements);
   cudaErrCheck(hipGetLastError());
   
   return 0;
}

// Fused forward kernel
__global__ void elementWise_fp(int hiddenSize, int miniBatch,
                               float *tmp_h, 
                               float *tmp_i, 
                               float *bias,
                               float *linearGates,
                               float *h_out,
                               float *i_out,
                               float *c_in,
                               float *c_out,
                               bool training) {
   int index = blockIdx.x * blockDim.x + threadIdx.x;
   int numElements = miniBatch * hiddenSize;
   
   if (index >= numElements) return;
   
   int batch = index / hiddenSize;
   int gateIndex = (index % hiddenSize) + 4 * batch * hiddenSize;   
   
   float g[4];

   for (int i = 0; i < 4; i++) {
      g[i] = tmp_i[i * hiddenSize + gateIndex] + tmp_h[i * hiddenSize + gateIndex];
      g[i] += bias[i * hiddenSize + index % hiddenSize] + bias[(i + 4) * hiddenSize + index % hiddenSize];
      
      if (training) linearGates[gateIndex + i * hiddenSize] = g[i];
   }   
   
   
   float in_gate     = sigmoidf(g[0]);
   float forget_gate = sigmoidf(g[1]);
   float in_gate2    = tanhf(g[2]);
   float out_gate    = sigmoidf(g[3]);
   
   float val = (forget_gate * c_in[index]) + (in_gate * in_gate2);
   
   c_out[index] = val;
   
   val = out_gate * tanhf(val);                                   

   h_out[index] = val;
   i_out[index] = val;
}


float LSTMTest(int hiddenSize, int miniBatch, int seqLength, int numLayers, bool checkF) {
   float *h_data;
   float *i_data;
   float *c_data;
      
   float *T;
   float *T_f;
   
   float *bias;
   
   float *tmp_h;
   float *tmp_i;
   float *linearGates;
   
   hipStream_t *stream_i;
   hipStream_t *stream_h;
   
   hipEvent_t **events_i;
   hipEvent_t **events_h;

   // Need a cuBLAS handle.
   hipblasHandle_t handle;
   cublasErrCheck(hipblasCreate(&handle));
   
   // Allocate streams/events
   stream_i = (hipStream_t*)malloc(numLayers * sizeof(hipStream_t));
   stream_h = (hipStream_t*)malloc(numLayers * sizeof(hipStream_t));
   
   // If we don't want to use streams we can launch everything in to the NULL stream
   for (int i = 0; i < numLayers; i++) {
      if (USE_STREAMS) {
         cudaErrCheck(hipStreamCreate(&stream_i[i]));
         // Priority is empirical.
         cudaErrCheck(hipStreamCreateWithPriority(&stream_h[i], 0, -1));   
      }
      else {
         stream_i[i] = NULL;  
         stream_h[i] = NULL;  
      }
   }
   
   
   events_i = (hipEvent_t**)malloc(numLayers * sizeof(hipEvent_t*));
   events_h = (hipEvent_t**)malloc(numLayers * sizeof(hipEvent_t*));
   for (int i = 0; i < numLayers; i++) {
      events_i[i] = (hipEvent_t*)malloc(seqLength * sizeof(hipEvent_t));
      events_h[i] = (hipEvent_t*)malloc(seqLength * sizeof(hipEvent_t));
   }
   
   
   // Input/output data
   int numElements = hiddenSize * miniBatch;
   
   cudaErrCheck(hipMalloc((void**)&h_data, (seqLength + 1) * (numLayers) * numElements * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&i_data, (seqLength) * (numLayers + 1) * numElements * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&c_data, (seqLength + 1) * (numLayers) * numElements * sizeof(float)));
   
   cudaErrCheck(hipMalloc((void**)&T, numLayers * hiddenSize * hiddenSize * 8 * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&T_f, numLayers * hiddenSize * hiddenSize * 8 * sizeof(float)));
   
   cudaErrCheck(hipMalloc((void**)&bias, numLayers * hiddenSize * 8 * sizeof(float)));
   
   // Workspace
   cudaErrCheck(hipMalloc((void**)&tmp_h, 4 * numLayers * numElements * sizeof(float)));
   cudaErrCheck(hipMalloc((void**)&tmp_i, 4 * seqLength * numElements * sizeof(float)));
   
   // Activations
   if (TRAINING) {
      cudaErrCheck(hipMalloc((void**)&linearGates, 4 * seqLength * numLayers * numElements * sizeof(float)));
   }
  

   // Initialise with random values.
   hiprandGenerator_t rng;
   curandErrCheck(hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT));
   curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(rng, 1337ull));
   curandErrCheck(hiprandGenerateUniform(rng, h_data, (seqLength + 1) * (numLayers) * numElements));
   curandErrCheck(hiprandGenerateUniform(rng, c_data, (seqLength + 1) * (numLayers) * numElements));
   curandErrCheck(hiprandGenerateUniform(rng, i_data, (seqLength) * (numLayers + 1) * numElements));
   
   curandErrCheck(hiprandGenerateUniform(rng, T, numLayers * hiddenSize * hiddenSize * 8));
   curandErrCheck(hiprandGenerateUniform(rng, bias, numLayers * hiddenSize * 8));
   curandErrCheck(hiprandDestroyGenerator(rng));

   
   // Make sure everything is done before we start the timers
   cudaErrCheck(hipDeviceSynchronize());
  
   // Timing starts here
   float elapsedTime;
   
   hipEvent_t start, stop;
   cudaErrCheck(hipEventCreate(&start));
   cudaErrCheck(hipEventCreate(&stop));

   cudaErrCheck(hipEventRecord(start));
  
   float alpha = 1.f;
   float beta  = 0.f;      
   
   const hipblasOperation_t transa = (PRE_TRANSPOSE && (seqLength > 1)) ? HIPBLAS_OP_N : HIPBLAS_OP_T;
   const hipblasOperation_t transb = HIPBLAS_OP_N;
   
   // Optimization 4
   if (transa == HIPBLAS_OP_N) {      
      for (int layer = 0; layer < numLayers; layer++) {                     
         float *T_i_in = T + layer * hiddenSize * hiddenSize * 8;
         float *T_i_out = T_f + layer * hiddenSize * hiddenSize * 8;

         float *T_h_in = T + layer * hiddenSize * hiddenSize * 8 + hiddenSize * hiddenSize * 4;
         float *T_h_out = T_f + layer * hiddenSize * hiddenSize * 8 + hiddenSize * hiddenSize * 4;

         cublasErrCheck(hipblasSetStream(handle, stream_i[layer]));
         cublasErrCheck(hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 4 * hiddenSize, hiddenSize, &alpha, T_i_in, hiddenSize, &beta, NULL, 4 * hiddenSize, T_i_out, 4 * hiddenSize));
         
         cublasErrCheck(hipblasSetStream(handle, stream_h[layer]));
         cublasErrCheck(hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, 4 * hiddenSize, hiddenSize, &alpha, T_h_in, hiddenSize, &beta, NULL, 4 * hiddenSize, T_h_out, 4 * hiddenSize));
      }      
   }
   else {
      T_f = T;
   }
   
   if (transb != HIPBLAS_OP_N) {
      printf("Only transb == HIPBLAS_OP_N supported\n");
      return -1;
   }
   
   int lStart = 0;
   int lEnd = 0;
   int rStart = 0;
   int rEnd = 0;
   
   int recurBatchSize = RECUR_BATCH_SIZE;
   
   while (true) {
      // Many layer "scheduling".
      if (lEnd == 0) {
         lStart = 0;
         lEnd = 1;
         rStart = 0;
      }
      else {
         // Move "up" and "left"
         lStart++;
         lEnd++;
         
         rStart -= recurBatchSize;
         
         // Over the top or off the left, reset to layer 0
         if (lEnd > numLayers || rStart < 0) {
            rStart += (lStart + 1) * recurBatchSize;

            lStart = 0;
            lEnd = 1;
         }
         
         // Off the right, step up
         while (rStart >= seqLength && lEnd <= numLayers) {
            lStart++;
            lEnd++;
            
            rStart -= recurBatchSize;
         }
         
         
         // Over the top or off the left, done!
         if (lEnd > numLayers || rStart < 0) {
            break;
         }
      }
      
      rEnd = rStart + recurBatchSize;
      if (rEnd > seqLength) rEnd = seqLength;
      
      for (int layer = lStart; layer < lEnd; layer++) {         
         cublasErrCheck(hipblasSetStream(handle, stream_i[layer]));
         
         for (int i = rStart; i < rEnd; i++) {
            if (layer > 0) {
               cudaErrCheck(hipStreamWaitEvent(stream_i[layer], events_h[layer - 1][i], 0));
               cudaErrCheck(hipEventDestroy(events_h[layer - 1][i]));
            }
         }

         // Optimization 1
         if (GROUP_GEMM) {
            cublasErrCheck(hipblasSgemm(handle,
                        transa, transb,
                        4 * hiddenSize, miniBatch * (rEnd - rStart), hiddenSize,
                        &alpha,
                        &T_f[layer * 8 * hiddenSize * hiddenSize],
                        transa == HIPBLAS_OP_N ? 4 * hiddenSize : hiddenSize,
                        i_data + rStart * numElements + layer * seqLength * numElements,
                        hiddenSize,
                        &beta,
                        tmp_i + 4 * rStart * numElements,
                        4 * hiddenSize));
         }
         else {
            for (int igemm =0; igemm < 4; igemm++) {
               cublasErrCheck(hipblasSgemm(handle,
                        transa, transb,
                        hiddenSize, miniBatch * (rEnd - rStart), hiddenSize,
                        &alpha,
                        &T_f[layer * 8 * hiddenSize * hiddenSize + igemm * hiddenSize],
                        transa == HIPBLAS_OP_N ? 4 * hiddenSize : hiddenSize,
                        i_data + rStart * numElements + layer * seqLength * numElements,
                        hiddenSize,
                        &beta,
                        tmp_i + 4 * rStart * numElements + igemm * hiddenSize,
                        4 * hiddenSize)); 
            }
         }
         
         for (int i = rStart; i < rEnd; i++) {
            cudaErrCheck(hipEventCreate(&events_i[layer][i], hipEventDisableTiming));
            cudaErrCheck(hipEventRecord(events_i[layer][i], stream_i[layer]));  
         }            
         
         for (int i = rStart; i < rEnd; i++) {
            cublasErrCheck(hipblasSetStream(handle, stream_h[layer]));
            // Optimization 1
            if (GROUP_GEMM) {
               cublasErrCheck(hipblasSgemm(handle,
                           transa, transb,
                           4 * hiddenSize, miniBatch, hiddenSize,
                           &alpha,
                           &T_f[4 * hiddenSize * hiddenSize + layer * 8 * hiddenSize * hiddenSize], 
                           transa == HIPBLAS_OP_N ? 4 * hiddenSize : hiddenSize,
                           h_data + i * numElements + layer * (seqLength + 1) * numElements,
                           hiddenSize,
                           &beta,
                           tmp_h + 4 * layer * numElements, 
                           4 * hiddenSize));
            }
            else {
               for (int igemm =0; igemm < 4; igemm++) {
                  cublasErrCheck(hipblasSgemm(handle,
                              transa, transb,
                              hiddenSize, miniBatch, hiddenSize,
                              &alpha,
                              &T_f[4 * hiddenSize * hiddenSize + layer * 8 * hiddenSize * hiddenSize + igemm * hiddenSize], 
                              transa == HIPBLAS_OP_N ? 4 * hiddenSize : hiddenSize,
                              h_data + i * numElements + layer * (seqLength + 1) * numElements,
                              hiddenSize,
                              &beta,
                              tmp_h + 4 * layer * numElements + igemm * hiddenSize, 
                              4 * hiddenSize));
               }
            }

            cudaErrCheck(hipStreamWaitEvent(stream_h[layer], events_i[layer][i], 0));
            cudaErrCheck(hipEventDestroy(events_i[layer][i]));

            // Optimization 3
            if (FUSE_PW) {
               dim3 blockDim;
               dim3 gridDim;
               
               blockDim.x = 256;
               gridDim.x = (numElements + blockDim.x - 1) / blockDim.x;               
               
               elementWise_fp <<< gridDim, blockDim , 0, stream_h[layer] >>> 
                      (hiddenSize, miniBatch,
                       tmp_h + 4 * layer * numElements, 
                       tmp_i + 4 * i * numElements, 
                       bias + 8 * layer * hiddenSize,
                       TRAINING ? linearGates + 4 * (i * numElements + layer * seqLength * numElements) : NULL,
                       h_data + (i + 1) * numElements + layer * (seqLength + 1) * numElements,
                       i_data + i * numElements + (layer + 1) * seqLength * numElements,
                       c_data + i * numElements + layer * (seqLength + 1) * numElements,
                       c_data + (i + 1) * numElements + layer * (seqLength + 1) * numElements,
                       TRAINING);
               cudaErrCheck(hipGetLastError());
            }
            else {
               LSTM_elementwise_unfused(hiddenSize, miniBatch,
                       tmp_h + 4 * layer * numElements, 
                       tmp_i + 4 * i * numElements, 
                       bias + 8 * layer * hiddenSize,
                       TRAINING ? linearGates + 4 * (i * numElements + layer * seqLength * numElements) : NULL,
                       h_data + (i + 1) * numElements + layer * (seqLength + 1) * numElements,
                       i_data + i * numElements + (layer + 1) * seqLength * numElements,
                       c_data + i * numElements + layer * (seqLength + 1) * numElements,
                       c_data + (i + 1) * numElements + layer * (seqLength + 1) * numElements,
                       TRAINING,
                       stream_h[layer]);
            }
            if (layer != numLayers - 1) {
               cudaErrCheck(hipEventCreate(&events_h[layer][i], hipEventDisableTiming));
               cudaErrCheck(hipEventRecord(events_h[layer][i], stream_h[layer]));  
            }
         }
      }
   } 
   cudaErrCheck(hipEventRecord(stop));
   cudaErrCheck(hipEventSynchronize(stop));
   cudaErrCheck(hipEventElapsedTime(&elapsedTime, start, stop));
   
   cudaErrCheck(hipDeviceSynchronize());

   
   // We're done. Print some checksums
   if (checkF) {
      float* testOutputi;
      float* testOutputh;
      float* testOutputc;
      
      
      int numElements = hiddenSize * miniBatch;
      testOutputi = (float*)malloc(numElements * seqLength * sizeof(float));
      testOutputh = (float*)malloc(numElements * numLayers * sizeof(float));
      testOutputc = (float*)malloc(numElements * numLayers * sizeof(float));
   
      cudaErrCheck(hipMemcpy(testOutputi, i_data + numLayers * seqLength * numElements, seqLength * numElements * sizeof(float), hipMemcpyDeviceToHost));
      for (int layer = 0; layer < numLayers; layer++) {
         cudaErrCheck(hipMemcpy(testOutputh + layer * numElements, h_data + seqLength * numElements + layer * (seqLength + 1) * numElements, numElements * sizeof(float), hipMemcpyDeviceToHost));
         cudaErrCheck(hipMemcpy(testOutputc + layer * numElements, c_data + seqLength * numElements + layer * (seqLength + 1) * numElements, numElements * sizeof(float), hipMemcpyDeviceToHost));
      }
      double checksumi = 0.;
      double checksumh = 0.;
      double checksumc = 0.;
      
      for (int m = 0; m < miniBatch; m++) {
         for (int j = 0; j < seqLength; j++) {
            for (int i = 0; i < hiddenSize; i++) {
               checksumi += testOutputi[j * numElements + m * hiddenSize + i];
               if (hiddenSize <= 8) printf("i: (%d,%d): %E\n", j, i, testOutputi[j * numElements + m * hiddenSize + i]);
            }
         }
         for (int j = 0; j < numLayers; j++) {
            for (int i = 0; i < hiddenSize; i++) {         
               checksumh += testOutputh[j * numElements + m * hiddenSize + i];
               checksumc += testOutputc[j * numElements + m * hiddenSize + i];
            }
         }
         
         if (m == 0) printf("i checksum (example %d) %E\n", m, checksumi);
         if (m == 0) printf("h checksum (example %d) %E\n", m, checksumh);
         if (m == 0) printf("c checksum (example %d) %E\n", m, checksumc);
      }
      
      printf("i checksum %E     ", checksumi);
      printf("c checksum %E     ", checksumc);
      printf("h checksum %E\n", checksumh);
      
      free(testOutputi);
      free(testOutputc);
      free(testOutputh);

   }
   
   cudaErrCheck(hipDeviceSynchronize());

   cudaErrCheck(hipFree(h_data));
   cudaErrCheck(hipFree(i_data));  
   cudaErrCheck(hipFree(c_data));  

   if (T != T_f) cudaErrCheck(hipFree(T)); 
   cudaErrCheck(hipFree(T_f));
   
   cudaErrCheck(hipFree(bias));
   
   cudaErrCheck(hipFree(tmp_h));
   cudaErrCheck(hipFree(tmp_i));
   if (TRAINING) cudaErrCheck(hipFree(linearGates));

   
   for (int i = 0; i < numLayers; i++) {
      if (stream_i[i] != NULL) cudaErrCheck(hipStreamDestroy(stream_i[i]));
      if (stream_h[i] != NULL) cudaErrCheck(hipStreamDestroy(stream_h[i]));
   }

   free(stream_i);
   free(stream_h);
   
   for (int i = 0; i < numLayers; i++) {
      free(events_i[i]);
      free(events_h[i]);
   }
   free(events_i);
   free(events_h);
   
   return elapsedTime;
}


int main(int argc, char* argv[]) {
   int seqLength;
   int numLayers;
   int hiddenSize;
   int miniBatch; 
   
   if (argc == 5) {
      seqLength = atoi(argv[1]);
      numLayers =  atoi(argv[2]);
      hiddenSize =  atoi(argv[3]);
      miniBatch =  atoi(argv[4]);   
   }
   else if (argc == 1) {
      printf("Running with default settings\n");
      seqLength = 100;
      numLayers = 4;
      hiddenSize = 512;
      miniBatch = 64;
   }
   else {
      printf("Usage: ./LSTM <seqLength> <numLayers> <hiddenSize> <miniBatch>\n");
      return 1;      
   }

   printf("seqLength %d, numLayers %d, hiddenSize %d, miniBatch %d\n", seqLength, numLayers, hiddenSize, miniBatch);  
   
   int numRuns = 1;
   
   float totalTime = 0.f;
   for (int run = 0; run < numRuns; run++) {
      totalTime += LSTMTest(hiddenSize, miniBatch, seqLength, numLayers, true);
   }
   
   printf("Runtime %fms\n", totalTime / numRuns);
   
   return time < 0;
}

