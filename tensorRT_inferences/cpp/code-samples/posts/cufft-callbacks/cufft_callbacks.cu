/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#include "common.h"

#define TILE_DIM 32
#define BLOCK_ROWS 8

////////////////////////////////////////////////////////////////////////////////
// Callback Implementations
////////////////////////////////////////////////////////////////////////////////
__device__ hipfftReal CB_ConvertInputR(void *dataIn, size_t offset, void *callerInfo, void *sharedPtr) {
    char element = ((char*)dataIn)[offset];
    return (hipfftReal)((float)element/127.0f);
}

__device__ hipfftCallbackLoadR d_loadCallbackPtr = CB_ConvertInputR; 

__device__ void CB_ConvolveAndStoreTransposedC(void *dataOut, size_t offset, hipfftComplex element, void *callerInfo, void *sharedPtr) {
    hipfftComplex *filter = (hipfftComplex*)callerInfo;
    size_t row = offset / COMPLEX_SIGNAL_SIZE;
    size_t col = offset % COMPLEX_SIGNAL_SIZE;

    ((hipfftComplex*)dataOut)[col * BATCH_SIZE + row] = ComplexMul(element, filter[col]);
}

__device__ hipfftCallbackStoreC d_storeCallbackPtr = CB_ConvolveAndStoreTransposedC;

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, const char **argv)
{
    struct hipDeviceProp_t properties;
    int device = argc > 1 ? atoi(argv[1]) : 0;

    checkCudaErrors(hipGetDevice(&device));
    checkCudaErrors(hipGetDeviceProperties(&properties, device));
    if( !(properties.major >= 2) ) {
        printf("This sample requires CUDA architecture SM2.0 or higher\n");
        exit(EXIT_FAILURE);
    }

    // Allocate and initialize memory
    printf("Preparing input: %dx%d\n", BATCH_SIZE, INPUT_SIGNAL_SIZE);
    char *_8bit_signal;
    hipfftComplex *result, *filter;

    checkCudaErrors(hipMallocManaged(&_8bit_signal, sizeof(char) * INPUT_SIGNAL_SIZE * BATCH_SIZE, hipMemAttachGlobal));
    checkCudaErrors(hipMallocManaged(&result, sizeof(hipfftComplex) * COMPLEX_SIGNAL_SIZE * BATCH_SIZE, hipMemAttachGlobal));
    checkCudaErrors(hipMallocManaged(&filter, sizeof(hipfftComplex) * COMPLEX_SIGNAL_SIZE, hipMemAttachGlobal));

    initInputs(_8bit_signal, filter);
    
    //compute reference result for later verification
    printf("Computing reference solution\n");
    hipfftComplex *reference = computeReference(_8bit_signal, filter);

    printf("Creating FFT plan\n");
    hipfftHandle fftPlan;
    size_t workSize;
    
    checkCudaErrors(hipfftCreate(&fftPlan));
    int signalSize = INPUT_SIGNAL_SIZE;
    checkCudaErrors(hipfftMakePlanMany(fftPlan, 1, &signalSize, 0,0,0,0,0,0, HIPFFT_R2C, BATCH_SIZE, &workSize));

    /*
     * Retrieve address of callback functions on the device
     */                              
    hipfftCallbackLoadR h_loadCallbackPtr;
    hipfftCallbackStoreC h_storeCallbackPtr;
    checkCudaErrors(hipMemcpyFromSymbol(&h_loadCallbackPtr, 
                                          HIP_SYMBOL(d_loadCallbackPtr), 
                                          sizeof(h_loadCallbackPtr)));
    checkCudaErrors(hipMemcpyFromSymbol(&h_storeCallbackPtr, 
                                          HIP_SYMBOL(d_storeCallbackPtr), 
                                          sizeof(h_storeCallbackPtr)));

    // Now associate the callbacks with the plan.
    hipfftResult status = hipfftXtSetCallback(fftPlan, 
                            (void **)&h_loadCallbackPtr, 
                            HIPFFT_CB_LD_REAL,
                            0);
    if (status == CUFFT_LICENSE_ERROR) {
        printf("This sample requires a valid license file.\n");
        printf("The file was either not found, out of date, or otherwise invalid.\n");
        exit(EXIT_FAILURE);
    } else {
        checkCudaErrors(status);
    }

   checkCudaErrors(hipfftXtSetCallback(fftPlan, 
                                (void **)&h_storeCallbackPtr, 
                                HIPFFT_CB_ST_COMPLEX,
                                (void **)&filter));

    //create timers
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    float elapsedTime;

    printf("Running %d iterations\n", ITERATIONS);
    checkCudaErrors(hipEventRecord(start, 0));

    /*
     * The actual Computation
     */

    for(int i = 0; i < ITERATIONS; i++) {
        checkCudaErrors(hipfftExecR2C(fftPlan, (hipfftReal*)_8bit_signal, result));
    }

    checkCudaErrors(hipEventRecord(end, 0));
    checkCudaErrors(hipEventSynchronize(end));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, end));
    printf("Time for the FFT: %fms\n", elapsedTime);

    //Verify correct result    
    if(postprocess(reference, result, COMPLEX_SIGNAL_SIZE * BATCH_SIZE)) {
        printf("Verification successful.\n");
    } else {
        printf("!!! Verification Failed !!!\n");
    }

    //Cleanup
    checkCudaErrors(hipfftDestroy(fftPlan));

    checkCudaErrors(hipFree(_8bit_signal));
    checkCudaErrors(hipFree(result));
    checkCudaErrors(hipFree(filter));
    checkCudaErrors(hipFree(reference));
      
    //clean up driver state
    hipDeviceReset();

    printf("Done\n");
    
    return 0;
}